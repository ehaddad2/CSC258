
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <chrono>

using namespace std;


__global__
void matrixMultCuda(int n, int* a, int* b, int* c)
{
	//each thread will calculate a single value in c
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = tid/n;
	if(i < n)
	{
		int j = tid%n;
		
		int total = 0;
		for(int k=0; k<n; k++)
		{
			total += a[i*n + k] * b[k*n + j];
		}
		c[tid] = total;
	}
}

int main(int argc, char* argv[])
{
	using chrono::high_resolution_clock;
	using chrono::duration;
	
	auto actualstart = high_resolution_clock::now();
	
	int n;
	cin >> n;
	int* m1;
	int* m2;
	int* m3;
	m1 = (int*)malloc(n*n*sizeof(int));
	m2 = (int*)malloc(n*n*sizeof(int));
	m3 = (int*)malloc(n*n*sizeof(int));
	
	for(int i=0; i<n; i++)
	{
		for(int j=0; j<n; j++)
		{
			cin >> m1[i*n + j];
		}
	}
	
	for(int i=0; i<n; i++)
	{
		for(int j=0; j<n; j++)
		{
			cin >> m2[i*n + j];
		}
	}
	
	auto start = high_resolution_clock::now();
	
	int* d_m1;
	int* d_m2;
	int* d_m3;
	
	hipMalloc(&d_m1, n*n*sizeof(int));
	hipMalloc(&d_m2, n*n*sizeof(int));
	hipMalloc(&d_m3, n*n*sizeof(int));
	
	hipMemcpy(d_m1, m1, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_m2, m2, n*n*sizeof(int), hipMemcpyHostToDevice);
	
	int dthreadcount = atoi(argv[1]);
	
	matrixMultCuda<<< (n*n + dthreadcount-1)/dthreadcount, dthreadcount>>>(n, d_m1, d_m2, d_m3);

	hipMemcpy(m3, d_m3, n*n*sizeof(int), hipMemcpyDeviceToHost);
	
	auto end = high_resolution_clock::now();
	
	duration<double, milli> partime = end - start;
	duration<double, milli> actualtime = end - actualstart;
	
	cout<<"Parallel time is: "<<partime.count()<<" milliseconds."<<endl;
	cout<<"Total time is: "<<actualtime.count()<<" milliseconds."<<endl;
	
	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_m3);
	free(m1);
	free(m2);
	free(m3);
}